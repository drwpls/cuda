#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <vector_functions.h>

#define WIDTH_REMOVE 50

#define FILTER_WIDTH 3
__constant__ int d_const_filterXSobel[FILTER_WIDTH * FILTER_WIDTH];
__constant__ int d_const_filterYSobel[FILTER_WIDTH * FILTER_WIDTH];

#define CHECK(call)                                                \
    {                                                              \
        const hipError_t error = call;                            \
        if (error != hipSuccess)                                  \
        {                                                          \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
            exit(EXIT_FAILURE);                                    \
        }                                                          \
    }

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char *fileName, int &width, int &height, uchar3 *&pixels)
{
    FILE *f = fopen(fileName, "r");
    if (f == NULL)
    {
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    char type[3];
    fscanf(f, "%s", type);

    if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
    {
        fclose(f);
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    fscanf(f, "%i", &width);
    fscanf(f, "%i", &height);

    int max_val;
    fscanf(f, "%i", &max_val);
    if (max_val > 255) // In this exercise, we assume 1 byte per value
    {
        fclose(f);
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
    for (int i = 0; i < width * height; i++)
        fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

    fclose(f);
}

void writePnm(uchar3 *pixels, int width, int height, char *fileName)
{
    FILE *f = fopen(fileName, "w");
    if (f == NULL)
    {
        printf("Cannot write %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    fprintf(f, "P3\n%i\n%i\n255\n", width, height);

    printf("\nImage output size (width x height): %i x %i\n", width, height);

    for (int i = 0; i < width * height; i++)
        fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);

    fclose(f);
}

void convertGrayscale(uchar3 *inPixels, int width, int height, uint8_t *grayPixels)
{
    for (int r = 0; r < height; r++)
    {
        for (int c = 0; c < width; c++)
        {
            int i = r * width + c;
            uint8_t red = inPixels[i].x;
            uint8_t green = inPixels[i].y;
            uint8_t blue = inPixels[i].z;
            grayPixels[i] = 0.299f * red + 0.587f * green + 0.114f * blue;
        }
    }
}

__global__ void convertGrayscaleKernel(uchar3 *inPixels, int width, int height, uint8_t *grayPixels)
{
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height && c < width)
    {
        int index = r * width + c;
        uchar3 inPixel = inPixels[index];
        uint8_t red = inPixel.x;
        uint8_t green = inPixel.y;
        uint8_t blue = inPixel.z;
        grayPixels[index] = 0.299f * red + 0.587f * green + 0.114f * blue;
    }
}

int abs(int x)
{
    return x < 0 ? -1 * x : x;
}

void calcEnergy(uint8_t *grayPixels, int width, int height, int *energyMap,
                int *filterXSobel, int *filterYSobel, int filterWidth)
{
    for (int r = 0; r < height; r++)
    {
        for (int c = 0; c < width; c++)
        {
            int convolutionX = 0;
            int convolutionY = 0;

            for (int filterR = 0; filterR < filterWidth; filterR++)
            {
                for (int filterC = 0; filterC < filterWidth; filterC++)
                {
                    // Calc convolution with X-Sobel filter
                    int filterValX = filterXSobel[filterR * filterWidth + filterC];
                    int grayPixelsR = r - filterWidth / 2 + filterR;
                    int grayPixelsC = c - filterWidth / 2 + filterC;
                    grayPixelsR = min(max(0, grayPixelsR), height - 1);
                    grayPixelsC = min(max(0, grayPixelsC), width - 1);
                    uint8_t grayPixel = grayPixels[grayPixelsR * width + grayPixelsC];
                    convolutionX += filterValX * (int)grayPixel;

                    // Calc convolution with Y-Sobel filter
                    int filterValY = filterYSobel[filterR * filterWidth + filterC];
                    grayPixelsR = r - filterWidth / 2 + filterR;
                    grayPixelsC = c - filterWidth / 2 + filterC;
                    grayPixelsR = min(max(0, grayPixelsR), height - 1);
                    grayPixelsC = min(max(0, grayPixelsC), width - 1);
                    grayPixel = grayPixels[grayPixelsR * width + grayPixelsC];
                    convolutionY += filterValY * (int)grayPixel;
                }
            }

            energyMap[r * width + c] = abs(convolutionX) + abs(convolutionY);
        }
    }
}

__global__ void printConstantFilterDEBUG()
{
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < FILTER_WIDTH && c < FILTER_WIDTH)
    {
        int i = r * FILTER_WIDTH + c;
        int X = d_const_filterXSobel[i];
        int Y = d_const_filterYSobel[i];
        printf("%i %i: %i %i \n", r, c, X, Y);
    }
}


__global__ void calcEnergyKernelMemOptimized(uint8_t *grayPixels, int width, int height, int *energyMap, int filterWidth)
{

    extern __shared__ uint8_t s_grayPixels[];

    int top_left_x = blockIdx.x * blockDim.x - filterWidth / 2;
    int top_left_y = blockIdx.y * blockDim.y - filterWidth / 2;
    int bottom_right_x = top_left_x + blockDim.x + filterWidth - 1;
    int bottom_right_y = top_left_y + blockDim.y + filterWidth - 1;
    int shared_width = bottom_right_x - top_left_x;
    int shared_height = bottom_right_y - top_left_y;

    int total_cell = shared_width * shared_height;
    int total_thread = blockDim.x * blockDim.y;

    int cell_per_thread = (total_cell + total_thread - 1) / total_thread;
    int thread_id = threadIdx.y * blockDim.x + threadIdx.x;

    for (int i = 0; i < cell_per_thread; i++)
    {
        int serial_index_in_shared = i * total_thread + thread_id;
        int x_in_shared_mem = serial_index_in_shared % shared_width;
        int y_in_shared_mem = serial_index_in_shared / shared_width;

        int x_in_global_mem = top_left_x + x_in_shared_mem;
        int y_in_global_mem = top_left_y + y_in_shared_mem;

        if (x_in_global_mem < 0)
            x_in_global_mem = 0;
        if (x_in_global_mem > width - 1)
            x_in_global_mem = width - 1;
        if (y_in_global_mem < 0)
            y_in_global_mem = 0;
        if (y_in_global_mem > height - 1)
            y_in_global_mem = height - 1;

        int index_in_global_mem = y_in_global_mem * width + x_in_global_mem;
        int index_in_shared_mem = y_in_shared_mem * shared_width + x_in_shared_mem;
        if (index_in_shared_mem < total_cell)
            s_grayPixels[index_in_shared_mem] = grayPixels[index_in_global_mem];
    }

    __syncthreads();


    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height && c < width)
    {
        int i = r * width + c;

        int convolutionX = 0;
        int convolutionY = 0;

        for (int filterR = 0; filterR < filterWidth; filterR++)
        {
            for (int filterC = 0; filterC < filterWidth; filterC++)
            {
                int filterValX = d_const_filterXSobel[filterR * filterWidth + filterC];

                int grayPixelsR = r - filterWidth / 2 + filterR;
                int grayPixelsC = c - filterWidth / 2 + filterC;
                grayPixelsR = min(max(0, grayPixelsR), height - 1);
                grayPixelsC = min(max(0, grayPixelsC), width - 1);

                int row_in_shared_mem = threadIdx.y + filterR;
                int col_in_shared_mem = threadIdx.x + filterC;

                // Calc convolution with X-Sobel filter
                uint8_t grayPixel = s_grayPixels[grayPixelsR * width + grayPixelsC];
                uint8_t grayPixel = s_grayPixels[row_in_shared_mem * (filterWidth + blockDim.x - 1) + col_in_shared_mem];
                convolutionX += filterValX * (int)grayPixel;

                // Calc convolution with Y-Sobel filter
                int filterValY = d_const_filterYSobel[filterR * filterWidth + filterC];
                convolutionY += filterValY * (int)grayPixel;
            }
        }

        energyMap[i] = abs(convolutionX) + abs(convolutionY);
    }
}

__global__ void calcEnergyKernel(uint8_t *grayPixels, int width, int height, int *energyMap,
                                 int *filterXSobel, int *filterYSobel, int filterWidth)
{
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height && c < width)
    {
        int i = r * width + c;

        int convolutionX = 0;
        int convolutionY = 0;

        for (int filterR = 0; filterR < filterWidth; filterR++)
        {
            for (int filterC = 0; filterC < filterWidth; filterC++)
            {
                // Calc convolution with X-Sobel filter
                int filterValX = filterXSobel[filterR * filterWidth + filterC];
                int grayPixelsR = r - filterWidth / 2 + filterR;
                int grayPixelsC = c - filterWidth / 2 + filterC;
                grayPixelsR = min(max(0, grayPixelsR), height - 1);
                grayPixelsC = min(max(0, grayPixelsC), width - 1);
                uint8_t grayPixel = grayPixels[grayPixelsR * width + grayPixelsC];
                convolutionX += filterValX * (int)grayPixel;

                // Calc convolution with Y-Sobel filter
                int filterValY = filterYSobel[filterR * filterWidth + filterC];
                grayPixelsR = r - filterWidth / 2 + filterR;
                grayPixelsC = c - filterWidth / 2 + filterC;
                grayPixelsR = min(max(0, grayPixelsR), height - 1);
                grayPixelsC = min(max(0, grayPixelsC), width - 1);
                grayPixel = grayPixels[grayPixelsR * width + grayPixelsC];
                convolutionY += filterValY * (int)grayPixel;
            }
        }

        energyMap[i] = abs(convolutionX) + abs(convolutionY);
    }
}

void findMinimumSeam(int *energyMap, int width, int height,
                     int *backtrack, int *L1, int *L2)
{
    memcpy(L1, energyMap, width * sizeof(int));

    for (int r = 1; r < height; r++)
    {
        for (int c = 0; c < width; c++)
        {
            int i = r * width + c;
            int idx;
            int energyMin = 1e9;

            for (int k = -1; k < 2; k++)
            {
                if ((c + k < 0) || (c + k == width))
                    continue;

                if (energyMin > L1[c + k])
                {
                    energyMin = L1[c + k];
                    idx = k;
                }
            }

            backtrack[i] = c + idx;
            L2[c] = energyMap[i] + energyMin;
        }
        memcpy(L1, L2, width * sizeof(int));

        // To debug
        // if (r == 1) {
        //     for (int c = 0; c < width; c++) {
        //         printf("\n%i\n", L1[c]);
        //     }
        // }
    }
}

__global__ void memcpyDevice2DeviceInt(int *dst, int *src, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        dst[i] = src[i];
    }
}

__device__ int bCount = 0;
volatile __device__ int bCount1 = 0;
volatile __device__ int bRowCount = 0; // To count the number of blocks of a row that completed the calculation L1

__global__ void findMinimumSeamKernel(int *energyMap, int width, int height,
                                      int *backtrack, volatile int *L1, volatile int *L2)
{
    __shared__ int bi;

    // Get the index bi that has the order
    if (threadIdx.x == 0)
    {
        bi = atomicAdd(&bCount, 1);
    }
    __syncthreads();

    int blockPerRow = (width - 1) / blockDim.x + 1;
    int r = bi / blockPerRow;
    int c = (bi % blockPerRow) * blockDim.x + threadIdx.x;

    if (r == 0)
    {
        // This block code like line "memcpy(L1, energyMap, width * sizeof(int));" in function findMinimumSeam
        if (c < width)
        {
            L1[c] = energyMap[c];
        }

        __syncthreads();

        if (threadIdx.x == 0)
        {
            while (bCount1 < bi)
            {
            }
            bCount1 += 1;
        }
    }
    else if (r < height)
    {
        while ((int)(bCount1 / blockPerRow) < r)
        {
        } // make sure previous rows complete the calculation

        if (r > 1)
        { // This block code like line "memcpy(L1, L2, width * sizeof(int));" in function findMinimumSeam
            if (c < width)
            {
                L1[c] = L2[c];
            }
        }
        __syncthreads();

        while (bCount1 < bi)
        {
        } // make sure only 1 block updates bCount1 at a time

        if (threadIdx.x == 0)
        {
            bRowCount += 1;
            __threadfence();

            if ((int)((bCount1 + 1) / blockPerRow) < r + 1)
            {
                // This condition to make sure this case:
                // If width > blockSize.x: blockPerRow > 1:
                //          The 1st, 2nd, 3rd, ... block of current row can update bCount1.
                //          However, the last block of current row can only update bCount1 after completing the calculation of that block
                // In case: blockPerRow == 1:
                //          Update bCount1 after the current block completes the calculation
                bCount1 += 1;
            }
        }

        // Wait for block (bi+1) to finish setting L1[c] = L2[c], because current block uses the firsst L1 of block (bi+1)
        while ((bi % blockPerRow + 2) > bRowCount && bRowCount < blockPerRow)
        {
        }

        if (c < width)
        {
            int i = r * width + c;
            int idx;
            int energyMin = 1e9;

            for (int k = -1; k < 2; k++)
            {
                if ((c + k < 0) || (c + k == width))
                    continue;

                if (energyMin > L1[c + k])
                {
                    energyMin = L1[c + k];
                    idx = k;
                }
            }

            backtrack[i] = c + idx;
            L2[c] = energyMap[i] + energyMin;
            // L2[c] = c;
        }
        __syncthreads();

        // If the current block is the last block of current row, update bCount1 and reset bRowCount
        if (threadIdx.x == 0 && (int)((bCount1 + 1) / blockPerRow) == r + 1)
        {
            bCount1 += 1;
            bRowCount = 0;
        }
    }
}

void deleteSeam(uchar3 *inPixels, int width, int height, uchar3 *outPixels,
                int *backtrack, int *L1, int *L2)
{
    int *seamPath = (int *)malloc(height * sizeof(int));
    int energyMin = 1e9, posMin;
    for (int i = 0; i < width; i++)
    {
        // printf("\n%i\n", L2[i]);
        if (energyMin > L2[i])
        {
            energyMin = L2[i];
            posMin = i;
        }
    }
    // printf("\n%i %i\n", energyMin, posMin);
    for (int r = height - 1; r >= 0; r--)
    {
        seamPath[r] = posMin;
        // printf("\n%i\n", seamPath[r]);
        posMin = backtrack[r * width + posMin];
    }

    for (int r = 0; r < height; r++)
    {
        for (int c = 0; c < width; c++)
        {
            if (c == seamPath[r])
                continue;

            int i = r * width + c;
            int _i = r * (width - 1) + c;

            if (c > seamPath[r])
                _i--;
            outPixels[_i] = inPixels[i];
        }
    }

    free(seamPath);
}

__global__ void deleteSeamKernel(uchar3 *inPixels, int width, int height, uchar3 *outPixels, int *seamPath)
{
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height && c < width && c != seamPath[r])
    {
        int i = r * width + c;
        int _i = r * (width - 1) + c;

        if (c > seamPath[r])
            _i--;

        outPixels[_i] = inPixels[i];
    }
}

void outputResult(uchar3 *dstPixels, int width, int height, uchar3 *srcPixels)
{
    for (int r = 0; r < height; r++)
    {
        for (int c = 0; c < width; c++)
        {
            int i = r * width + c;
            dstPixels[i] = srcPixels[i];
        }
    }
}

void seamCarving(uchar3 *inPixels, int width, int height, uchar3 *outPixels,
                 int *filterXSobel, int *filterYSobel, int filterWidth,
                 bool useDevice = false, dim3 blockSize = dim3(1, 1), int kernelType = 1)
{
    GpuTimer timer;
    timer.Start();

    if (useDevice == false)
    {
        printf("\nSeam Carving by host\n");

        uint8_t *grayPixels = (uint8_t *)malloc(width * height * sizeof(uint8_t));
        int *energyMap = (int *)malloc(width * height * sizeof(int));
        int *backtrack = (int *)malloc(width * height * sizeof(int));
        int *L1 = (int *)malloc(width * sizeof(int));
        int *L2 = (int *)malloc(width * sizeof(int));

        uchar3 *tempPixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
        memcpy(tempPixels, inPixels, width * height * sizeof(uchar3));

        for (int i = 0; i < WIDTH_REMOVE; i++)
        {
            convertGrayscale(tempPixels, width - i, height, grayPixels);

            calcEnergy(grayPixels, width - i, height, energyMap, filterXSobel, filterYSobel, filterWidth);

            findMinimumSeam(energyMap, width - i, height, backtrack, L1, L2);

            deleteSeam(tempPixels, width - i, height, tempPixels, backtrack, L1, L2);
        }

        for (int r = 0; r < height; r++)
        {
            for (int c = 0; c < width - WIDTH_REMOVE; c++)
            {
                int i = r * (width - WIDTH_REMOVE) + c;
                outPixels[i] = tempPixels[i];
            }
        }

        free(grayPixels);
        free(energyMap);
        free(backtrack);
        free(L1);
        free(L2);
        free(tempPixels);
    }
    else // Use device
    {
        printf("\nSeam Carving by device\n");
        int zero = 0;

        uint8_t *d_grayPixels;
        int *d_energyMap, *d_backtrack, *d_L1, *d_L2;
        uchar3 *d_tempPixels, *d_tempPixels1;

        int *d_seamPath;

        CHECK(hipMalloc(&d_grayPixels, width * height * sizeof(uint8_t)));
        CHECK(hipMalloc(&d_energyMap, width * height * sizeof(int)));
        CHECK(hipMalloc(&d_backtrack, width * height * sizeof(int)));

        CHECK(hipMalloc(&d_L1, width * sizeof(int)));
        CHECK(hipMalloc(&d_L2, width * sizeof(int)));

        CHECK(hipMalloc(&d_tempPixels, width * height * sizeof(uchar3)));
        CHECK(hipMalloc(&d_tempPixels1, width * height * sizeof(uchar3)));
        CHECK(hipMemcpy(d_tempPixels, inPixels, width * height * sizeof(uchar3), hipMemcpyHostToDevice));

        CHECK(hipMalloc(&d_seamPath, height * sizeof(int)));

        if (kernelType == 1)
        {
            int *d_filterXSobel, *d_filterYSobel;
            CHECK(hipMalloc(&d_filterXSobel, filterWidth * filterWidth * sizeof(int)));
            CHECK(hipMalloc(&d_filterYSobel, filterWidth * filterWidth * sizeof(int)));

            CHECK(hipMemcpy(d_filterXSobel, filterXSobel, filterWidth * filterWidth * sizeof(int), hipMemcpyHostToDevice));
            CHECK(hipMemcpy(d_filterYSobel, filterYSobel, filterWidth * filterWidth * sizeof(int), hipMemcpyHostToDevice));

            for (int i = 0; i < WIDTH_REMOVE; i++)
            {
                dim3 gridSizeBlock2D((width - i - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
                int blockSize1D = (int)blockSize.x * (int)blockSize.y; // will be explained before use
                while (blockSize1D > 2 * width - 1)
                {
                    blockSize1D /= 2;
                }
                dim3 gridSizeBlock1D(((width - i - 1) / blockSize1D + 1) * height);

                // printf("\nblock size: %i , grid size: %i\n", blockSize1D, (int)gridSizeBlock1D.x);

                // convertGrayscaleKernel<<<gridSizeBlock2D, blockSize>>>(d_tempPixels, width - i, height, d_grayPixels);
                // hipDeviceSynchronize();
                // CHECK(hipGetLastError());

                // We can convert to grayscale by kernel, but output may be in some places different from output of host.
                // That leads to skewed results.
                // So we convert to grayscale by host.
                uchar3 *tempPixels = (uchar3 *)malloc((width - i) * height * sizeof(uchar3));
                uint8_t *grayPixels = (uint8_t *)malloc((width - i) * height * sizeof(uint8_t));

                CHECK(hipMemcpy(tempPixels, d_tempPixels, (width - i) * height * sizeof(uchar3), hipMemcpyDeviceToHost));

                convertGrayscale(tempPixels, width - i, height, grayPixels);

                CHECK(hipMemcpy(d_grayPixels, grayPixels, (width - i) * height * sizeof(uint8_t), hipMemcpyHostToDevice));

                free(tempPixels);
                free(grayPixels);

                calcEnergyKernel<<<gridSizeBlock2D, blockSize>>>(d_grayPixels, width - i, height, d_energyMap,
                                                                 d_filterXSobel, d_filterYSobel, filterWidth);
                hipDeviceSynchronize();
                CHECK(hipGetLastError());

                // Because we can only compute row by row sequentially, we have to use 1-dimensional block
                // We set blockSize1D = blockSize2D.x * blockSize2D.y to utilize resources (blockSize2D = blockSize)
                findMinimumSeamKernel<<<gridSizeBlock1D, blockSize1D>>>(d_energyMap, width - i, height,
                                                                        d_backtrack, d_L1, d_L2);
                hipDeviceSynchronize();
                CHECK(hipGetLastError());

                // Exact seam path
                int energyMin = 1e9;
                int *posMin = (int *)malloc(sizeof(int));
                int *curL2 = (int *)malloc(sizeof(int));

                for (int k = 0; k < width - i; k++)
                {
                    CHECK(hipMemcpy(curL2, &d_L2[k], sizeof(int), hipMemcpyDeviceToHost));
                    // printf("\n%i\n", curL2[0]);
                    if (energyMin > curL2[0])
                    {
                        energyMin = curL2[0];
                        posMin[0] = k;
                    }
                }
                // printf("\n%i %i\n", energyMin, posMin[0]);
                for (int r = height - 1; r >= 0; r--)
                {
                    CHECK(hipMemcpy(&d_seamPath[r], posMin, sizeof(int), hipMemcpyHostToDevice));
                    // printf("\n%i\n", posMin[0]);
                    CHECK(hipMemcpy(posMin, &d_backtrack[r * (width - i) + posMin[0]], sizeof(int), hipMemcpyDeviceToHost));
                }
                // End exact seam path

                free(posMin);
                free(curL2);

                deleteSeamKernel<<<gridSizeBlock2D, blockSize>>>(d_tempPixels, width - i, height, d_tempPixels1, d_seamPath);
                hipDeviceSynchronize();
                CHECK(hipGetLastError());

                uchar3 *temp = d_tempPixels;
                d_tempPixels = d_tempPixels1;
                d_tempPixels1 = temp;

                CHECK(hipMemcpyToSymbol(HIP_SYMBOL(bCount), &zero, sizeof(int)));
                CHECK(hipMemcpyToSymbol(HIP_SYMBOL(bCount1), &zero, sizeof(int)));
            }

            CHECK(hipFree(d_filterXSobel));
            CHECK(hipFree(d_filterYSobel));
        }

        if (kernelType == 2) {
            CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_const_filterXSobel), filterXSobel, FILTER_WIDTH * FILTER_WIDTH *sizeof(int)));
            CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_const_filterYSobel), filterYSobel, FILTER_WIDTH * FILTER_WIDTH *sizeof(int)));
            
            #ifdef DEBUG
            printConstantFilterDEBUG<<<1, 1>>>();
            #endif

            for (int i = 0; i < WIDTH_REMOVE; i++)
            {
                dim3 gridSizeBlock2D((width - i - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
                int blockSize1D = (int)blockSize.x * (int)blockSize.y; // will be explained before use
                while (blockSize1D > 2 * width - 1)
                {
                    blockSize1D /= 2;
                }
                dim3 gridSizeBlock1D(((width - i - 1) / blockSize1D + 1) * height);

                // printf("\nblock size: %i , grid size: %i\n", blockSize1D, (int)gridSizeBlock1D.x);

                // convertGrayscaleKernel<<<gridSizeBlock2D, blockSize>>>(d_tempPixels, width - i, height, d_grayPixels);
                // hipDeviceSynchronize();
                // CHECK(hipGetLastError());

                // We can convert to grayscale by kernel, but output may be in some places different from output of host.
                // That leads to skewed results.
                // So we convert to grayscale by host.
                uchar3 *tempPixels = (uchar3 *)malloc((width - i) * height * sizeof(uchar3));
                uint8_t *grayPixels = (uint8_t *)malloc((width - i) * height * sizeof(uint8_t));

                CHECK(hipMemcpy(tempPixels, d_tempPixels, (width - i) * height * sizeof(uchar3), hipMemcpyDeviceToHost));

                convertGrayscale(tempPixels, width - i, height, grayPixels);

                CHECK(hipMemcpy(d_grayPixels, grayPixels, (width - i) * height * sizeof(uint8_t), hipMemcpyHostToDevice));

                free(tempPixels);
                free(grayPixels);

                calcEnergyKernelMemOptimized<<<gridSizeBlock2D, blockSize, (blockSize.x + filterWidth - 1) * (blockSize.y + filterWidth - 1) * sizeof(uint8_t)>>>(d_grayPixels, width - i, height, d_energyMap, filterWidth);
                hipDeviceSynchronize();
                CHECK(hipGetLastError());

                // Because we can only compute row by row sequentially, we have to use 1-dimensional block
                // We set blockSize1D = blockSize2D.x * blockSize2D.y to utilize resources (blockSize2D = blockSize)
                findMinimumSeamKernel<<<gridSizeBlock1D, blockSize1D>>>(d_energyMap, width - i, height,
                                                                        d_backtrack, d_L1, d_L2);
                hipDeviceSynchronize();
                CHECK(hipGetLastError());

                // Exact seam path
                int energyMin = 1e9;
                int *posMin = (int *)malloc(sizeof(int));
                int *curL2 = (int *)malloc(sizeof(int));

                for (int k = 0; k < width - i; k++)
                {
                    CHECK(hipMemcpy(curL2, &d_L2[k], sizeof(int), hipMemcpyDeviceToHost));
                    // printf("\n%i\n", curL2[0]);
                    if (energyMin > curL2[0])
                    {
                        energyMin = curL2[0];
                        posMin[0] = k;
                    }
                }
                // printf("\n%i %i\n", energyMin, posMin[0]);
                for (int r = height - 1; r >= 0; r--)
                {
                    CHECK(hipMemcpy(&d_seamPath[r], posMin, sizeof(int), hipMemcpyHostToDevice));
                    // printf("\n%i\n", posMin[0]);
                    CHECK(hipMemcpy(posMin, &d_backtrack[r * (width - i) + posMin[0]], sizeof(int), hipMemcpyDeviceToHost));
                }
                // End exact seam path

                free(posMin);
                free(curL2);

                deleteSeamKernel<<<gridSizeBlock2D, blockSize>>>(d_tempPixels, width - i, height, d_tempPixels1, d_seamPath);
                hipDeviceSynchronize();
                CHECK(hipGetLastError());

                uchar3 *temp = d_tempPixels;
                d_tempPixels = d_tempPixels1;
                d_tempPixels1 = temp;

                CHECK(hipMemcpyToSymbol(HIP_SYMBOL(bCount), &zero, sizeof(int)));
                CHECK(hipMemcpyToSymbol(HIP_SYMBOL(bCount1), &zero, sizeof(int)));
            }

        }
        
        
        CHECK(hipMemcpy(outPixels, d_tempPixels, (width - WIDTH_REMOVE) * height * sizeof(uchar3), hipMemcpyDeviceToHost));

        CHECK(hipFree(d_grayPixels));
        CHECK(hipFree(d_energyMap));
        CHECK(hipFree(d_backtrack));
        CHECK(hipFree(d_L1));
        CHECK(hipFree(d_L2));
        CHECK(hipFree(d_tempPixels));
        CHECK(hipFree(d_tempPixels1));
        
        CHECK(hipFree(d_seamPath));
    }

    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
}

float computeError(uchar3 *a1, uchar3 *a2, int n)
{
    float err = 0;
    for (int i = 0; i < n; i++)
    {
        err += abs((int)a1[i].x - (int)a2[i].x);
        err += abs((int)a1[i].y - (int)a2[i].y);
        err += abs((int)a1[i].z - (int)a2[i].z);
    }
    err /= (n * 3);
    return err;
}

void printError(uchar3 *deviceResult, uchar3 *hostResult, int width, int height)
{
    float err = computeError(deviceResult, hostResult, width * height);
    printf("Error: %f\n", err);
}

char *concatStr(const char *s1, const char *s2)
{
    char *result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor);
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("CMEM: %lu bytes\n", devProv.totalConstMem);
    printf("L2 cache: %i bytes\n", devProv.l2CacheSize);
    printf("SMEM / one SM: %lu bytes\n", devProv.sharedMemPerMultiprocessor);

    printf("****************************\n");
}

int main(int argc, char **argv)
{
    if (argc != 3 && argc != 5)
    {
        printf("The number of arguments is invalid\n");
        return EXIT_FAILURE;
    }

    printDeviceInfo();

    // Read input image file
    int width, height;
    uchar3 *inPixels;
    readPnm(argv[1], width, height, inPixels);
    printf("\nImage size (width x height): %i x %i\n", width, height);

    // Set up a simple filter with blurring effect
    int filterWidth = FILTER_WIDTH;
    int *filterXSobel = (int *)malloc(filterWidth * filterWidth * sizeof(int));
    int *filterYSobel = (int *)malloc(filterWidth * filterWidth * sizeof(int));

    filterXSobel[0] = 1, filterXSobel[1] = 0, filterXSobel[2] = -1;
    filterXSobel[3] = 2, filterXSobel[4] = 0, filterXSobel[5] = -2;
    filterXSobel[6] = 1, filterXSobel[7] = 0, filterXSobel[8] = -1;

    filterYSobel[0] = 1, filterYSobel[1] = 2, filterYSobel[2] = 1;
    filterYSobel[3] = 0, filterYSobel[4] = 0, filterYSobel[5] = 0;
    filterYSobel[6] = -1, filterYSobel[7] = -2, filterYSobel[8] = -1;

    // Blur input image not using device
    uchar3 *outPixels = (uchar3 *)malloc((width - WIDTH_REMOVE) * height * sizeof(uchar3));
    seamCarving(inPixels, width, height, outPixels, filterXSobel, filterYSobel, filterWidth);

    // Blur input image using device, kernel 1
    dim3 blockSize(32, 32); // Default
    if (argc == 5)
    {
        blockSize.x = atoi(argv[3]);
        blockSize.y = atoi(argv[4]);
    }
    uchar3 *outPixels1 = (uchar3 *)malloc((width - (int)WIDTH_REMOVE) * height * sizeof(uchar3));
    seamCarving(inPixels, width, height, outPixels1, filterXSobel, filterYSobel, filterWidth, true, blockSize, 1);
    printError(outPixels1, outPixels, width - WIDTH_REMOVE, height);

    uchar3 *outPixels2 = (uchar3 *)malloc((width - (int)WIDTH_REMOVE) * height * sizeof(uchar3));
    seamCarving(inPixels, width, height, outPixels2, filterXSobel, filterYSobel, filterWidth, true, blockSize, 2);
    printError(outPixels2, outPixels, width - WIDTH_REMOVE, height);

    // Write results to files
    char *outFileNameBase = strtok(argv[2], "."); // Get rid of extension
    writePnm(outPixels, width - WIDTH_REMOVE, height, concatStr(outFileNameBase, "_host.pnm"));
    writePnm(outPixels1, width - WIDTH_REMOVE, height, concatStr(outFileNameBase, "_device1.pnm"));
    writePnm(outPixels2, width - WIDTH_REMOVE, height, concatStr(outFileNameBase, "_device2.pnm"));
    // writePnm(outPixels3, width, height, concatStr(outFileNameBase, "_device3.pnm"));

    // Free memories
    free(inPixels);
    free(filterXSobel);
    free(filterYSobel);
    free(outPixels);
    free(outPixels1);
    free(outPixels2);

}
