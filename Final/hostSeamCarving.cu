#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <vector_functions.h>

#define WIDTH_REMOVE 16

#define FILTER_WIDTH 3
// __constant__ float dc_filter[FILTER_WIDTH * FILTER_WIDTH];

#define CHECK(call)                                                \
    {                                                              \
        const hipError_t error = call;                            \
        if (error != hipSuccess)                                  \
        {                                                          \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
            exit(EXIT_FAILURE);                                    \
        }                                                          \
    }

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char *fileName, int &width, int &height, uchar3 *&pixels)
{
    FILE *f = fopen(fileName, "r");
    if (f == NULL)
    {
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    char type[3];
    fscanf(f, "%s", type);

    if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
    {
        fclose(f);
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    fscanf(f, "%i", &width);
    fscanf(f, "%i", &height);

    int max_val;
    fscanf(f, "%i", &max_val);
    if (max_val > 255) // In this exercise, we assume 1 byte per value
    {
        fclose(f);
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
    for (int i = 0; i < width * height; i++)
        fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

    fclose(f);
}

void writePnm(uchar3 *pixels, int width, int height, char *fileName)
{
    FILE *f = fopen(fileName, "w");
    if (f == NULL)
    {
        printf("Cannot write %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    fprintf(f, "P3\n%i\n%i\n255\n", width, height);

    printf("\nImage output size (width x height): %i x %i\n", width, height);

    for (int i = 0; i < width * height; i++)
        fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);

    fclose(f);
}

void convertGrayscale(uchar3 *inPixels, int width, int height, uint8_t * grayPixels)
{
    for (int r = 0; r < height; r++) {
        for (int c = 0; c < width; c++) {
            int i = r * width + c;
            uint8_t red = inPixels[i].x;
            uint8_t green = inPixels[i].y;
            uint8_t blue = inPixels[i].z;
            grayPixels[i] = 0.299f*red + 0.587f*green + 0.114f*blue;
        }
    }
}

int abs(int x) {
    return x < 0 ? -1 * x : x;
}

void calcEnergy(uint8_t * grayPixels, int width, int height, int * energyMap,
                int * filterXSobel, int * filterYSobel, int filterWidth)
{
    for (int r = 0; r < height; r++) {
        for (int c = 0; c < width; c++) {
            int convolutionX = 0;
            int convolutionY = 0;

            for (int filterR = 0; filterR < filterWidth; filterR++) {
                for (int filterC = 0; filterC < filterWidth; filterC++) {
                    // Calc convolution with X-Sobel filter
                    int filterValX = filterXSobel[filterR * filterWidth + filterC];
                    int grayPixelsR = r - filterWidth / 2 + filterR;
                    int grayPixelsC = c - filterWidth / 2 + filterC;
                    grayPixelsR = min(max(0, grayPixelsR), height - 1);
                    grayPixelsC = min(max(0, grayPixelsC), width - 1);
                    uint8_t grayPixel = grayPixels[grayPixelsR * width + grayPixelsC];
                    convolutionX += filterValX * (int)grayPixel;

                    // Calc convolution with Y-Sobel filter
                    int filterValY = filterYSobel[filterR * filterWidth + filterC];
                    grayPixelsR = r - filterWidth / 2 + filterR;
                    grayPixelsC = c - filterWidth / 2 + filterC;
                    grayPixelsR = min(max(0, grayPixelsR), height - 1);
                    grayPixelsC = min(max(0, grayPixelsC), width - 1);
                    grayPixel = grayPixels[grayPixelsR * width + grayPixelsC];
                    convolutionY += filterValY * (int)grayPixel;
                }
            }

            energyMap[r * width + c] = abs(convolutionX) + abs(convolutionY);
        }
    }
}

void findMinimumSeam(int * energyMap, int width, int height,
                    int * backtrack, int * L1, int * L2)
{
    memcpy(L1, energyMap, width * sizeof(int));

    for (int r = 1; r < height; r++) {
        for (int c = 0; c < width; c++) {
            int i = r * width + c;
            int idx;
            int energyMin = 1e9;
            
            for (int k = -1; k < 2; k++) {
                if ((c + k < 0) || (c + k == width))
                    continue;

                if (energyMin > L1[c + k]) {
                    energyMin = L1[c + k];
                    idx = k;
                }
            }

            backtrack[i] = c + idx;
            L2[c] = energyMap[i] + L1[c + idx];
        }
        memcpy(L1, L2, width * sizeof(int));
    }
}

void deleteSeam(uchar3 * inPixels, int width, int height, uchar3 * outPixels,
                int * backtrack, int * L1, int * L2)
{
    int * seamPath = (int *)malloc(height * sizeof(int));
    int energyMin = 1e9, posMin;
    for (int i = 0; i < width; i++) {
        if (energyMin > L2[i]) {
            energyMin = L2[i];
            posMin = i;
        }
    }
    for (int r = height - 1; r >= 0; r--) {
        seamPath[r] = posMin;
        // printf("\n%i\n", seamPath[r]);
        posMin = backtrack[r * width + posMin];
    }

    for (int r = 0; r < height; r++) {
        for (int c = 0; c < width; c++) {
            int i = r * width + c;
            int _i = r * (width - 1) + c;
            if (c == seamPath[r])
                continue;
            if (c > seamPath[r])
                _i--;
            outPixels[_i] = inPixels[i];
        }
    }

    free(seamPath);
}

void seamCarving(uchar3 * inPixels, int width, int height, uchar3 * outPixels,
                int * filterXSobel, int * filterYSobel, int filterWidth,
                bool useDevice = false, dim3 blockSize = dim3(1, 1), int kernelType = 1)
{
    if (useDevice == false)
    {
        uint8_t * grayPixels = (uint8_t *)malloc(width * height * sizeof(uint8_t));
        int * energyMap = (int *)malloc(width * height * sizeof(int));
        int * backtrack = (int *)malloc(width * height * sizeof(int));
        int * L1 = (int *)malloc(width * sizeof(int));
        int * L2 = (int *)malloc(width * sizeof(int));

        uchar3 * tempPixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
        memcpy(tempPixels, inPixels, width * height * sizeof(uchar3));

        for (int i = 0; i < WIDTH_REMOVE; i++) {
            convertGrayscale(tempPixels, width - i, height, grayPixels);

            calcEnergy(grayPixels, width - i, height, energyMap, filterXSobel, filterYSobel, filterWidth);

            findMinimumSeam(energyMap, width - i, height, backtrack, L1, L2);

            deleteSeam(tempPixels, width - i, height, tempPixels, backtrack, L1, L2);
        }

        for (int r = 0; r < height; r++) {
            for (int c = 0; c < width - WIDTH_REMOVE; c++) {
                int i = r * (width - WIDTH_REMOVE) + c;
                outPixels[i] = tempPixels[i];
            }
        }

        free(grayPixels);
        free(energyMap);
        free(backtrack);
        free(L1);
        free(L2);
        free(tempPixels);
    }
    else // Use device
    {
    //     GpuTimer timer;

        
    //     timer.Stop();
    //     float time = timer.Elapsed();
    //     printf("Kernel time: %f ms\n", time);
    //     hipDeviceSynchronize();
    //     CHECK(hipGetLastError());

    //     // Copy result from device memory
    //     CHECK(hipMemcpy(outPixels, d_outPixels, pixelsSize, hipMemcpyDeviceToHost));

    //     // Free device memories
    //     CHECK(hipFree(d_inPixels));
    //     CHECK(hipFree(d_outPixels));
    //     if (kernelType == 1 || kernelType == 2)
    //     {
    //         CHECK(hipFree(d_filter));
    //     }
    }
}

float computeError(uchar3 *a1, uchar3 *a2, int n)
{
    float err = 0;
    for (int i = 0; i < n; i++)
    {
        err += abs((int)a1[i].x - (int)a2[i].x);
        err += abs((int)a1[i].y - (int)a2[i].y);
        err += abs((int)a1[i].z - (int)a2[i].z);
    }
    err /= (n * 3);
    return err;
}

void printError(uchar3 *deviceResult, uchar3 *hostResult, int width, int height)
{
    float err = computeError(deviceResult, hostResult, width * height);
    printf("Error: %f\n", err);
}

char *concatStr(const char *s1, const char *s2)
{
    char *result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor);
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("CMEM: %lu bytes\n", devProv.totalConstMem);
    printf("L2 cache: %i bytes\n", devProv.l2CacheSize);
    printf("SMEM / one SM: %lu bytes\n", devProv.sharedMemPerMultiprocessor);

    printf("****************************\n");
}

int main(int argc, char **argv)
{
    if (argc != 3 && argc != 5)
    {
        printf("The number of arguments is invalid\n");
        return EXIT_FAILURE;
    }

    printDeviceInfo();

    // Read input image file
    int width, height;
    uchar3 *inPixels;
    readPnm(argv[1], width, height, inPixels);
    printf("\nImage size (width x height): %i x %i\n", width, height);

    // Set up a simple filter with blurring effect
    int filterWidth = FILTER_WIDTH;
    int *filterXSobel = (int *)malloc(filterWidth * filterWidth * sizeof(int));
    int *filterYSobel = (int *)malloc(filterWidth * filterWidth * sizeof(int));
    
    filterXSobel[0] = 1, filterXSobel[1] = 0, filterXSobel[2] = -1;
    filterXSobel[3] = 2, filterXSobel[4] = 0, filterXSobel[5] = -2;
    filterXSobel[6] = 1, filterXSobel[7] = 0, filterXSobel[8] = -1;

    filterYSobel[0] = 1, filterYSobel[1] = 2, filterYSobel[2] = 1;
    filterYSobel[3] = 0, filterYSobel[4] = 0, filterYSobel[5] = 0;
    filterYSobel[6] = -1, filterYSobel[7] = -2, filterYSobel[8] = -1;

    // Blur input image not using device
    uchar3 *outPixels = (uchar3 *)malloc((width - WIDTH_REMOVE) * height * sizeof(uchar3));
    seamCarving(inPixels, width, height, outPixels, filterXSobel, filterYSobel, filterWidth);

    // Blur input image using device, kernel 1
    // dim3 blockSize(32, 32); // Default
    // if (argc == 5)
    // {
    //     blockSize.x = atoi(argv[3]);
    //     blockSize.y = atoi(argv[4]);
    // }
    // uchar3 *outPixels1 = (uchar3 *)malloc(width * height * sizeof(uchar3));
    // blurImg(inPixels, width, height, filter, filterWidth, outPixels1, true, blockSize, 1);
    // printError(outPixels1, correctOutPixels, width, height);

    // Write results to files
    char *outFileNameBase = strtok(argv[2], "."); // Get rid of extension 
    writePnm(outPixels, width - WIDTH_REMOVE, height, concatStr(outFileNameBase, "_host.pnm"));
    // writePnm(outPixels1, width, height, concatStr(outFileNameBase, "_device1.pnm"));
    // writePnm(outPixels2, width, height, concatStr(outFileNameBase, "_device2.pnm"));
    // writePnm(outPixels3, width, height, concatStr(outFileNameBase, "_device3.pnm"));

    // Free memories
    free(inPixels);
    free(filterXSobel);
    free(filterYSobel);
    free(outPixels);
    // free(outPixels1);
}
