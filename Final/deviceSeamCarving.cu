#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <vector_functions.h>

#define FILTER_WIDTH 3
// __constant__ float dc_filter[FILTER_WIDTH * FILTER_WIDTH];

#define CHECK(call)                                                \
    {                                                              \
        const hipError_t error = call;                            \
        if (error != hipSuccess)                                  \
        {                                                          \
            fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
            fprintf(stderr, "code: %d, reason: %s\n", error,       \
                    hipGetErrorString(error));                    \
            exit(EXIT_FAILURE);                                    \
        }                                                          \
    }

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void readPnm(char *fileName, int &width, int &height, uchar3 *&pixels)
{
    FILE *f = fopen(fileName, "r");
    if (f == NULL)
    {
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    char type[3];
    fscanf(f, "%s", type);

    if (strcmp(type, "P3") != 0) // In this exercise, we don't touch other types
    {
        fclose(f);
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    fscanf(f, "%i", &width);
    fscanf(f, "%i", &height);

    int max_val;
    fscanf(f, "%i", &max_val);
    if (max_val > 255) // In this exercise, we assume 1 byte per value
    {
        fclose(f);
        printf("Cannot read %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    pixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
    for (int i = 0; i < width * height; i++)
        fscanf(f, "%hhu%hhu%hhu", &pixels[i].x, &pixels[i].y, &pixels[i].z);

    fclose(f);
}

void writePnm(uchar3 *pixels, int width, int height, char *fileName)
{
    FILE *f = fopen(fileName, "w");
    if (f == NULL)
    {
        printf("Cannot write %s\n", fileName);
        exit(EXIT_FAILURE);
    }

    fprintf(f, "P3\n%i\n%i\n255\n", width, height);

    printf("\nImage output size (width x height): %i x %i\n", width, height);

    for (int i = 0; i < width * height; i++)
        fprintf(f, "%hhu\n%hhu\n%hhu\n", pixels[i].x, pixels[i].y, pixels[i].z);

    fclose(f);
}

void convertGrayscale(uchar3 *inPixels, int width, int height, uint8_t * grayPixels)
{
    for (int r = 0; r < height; r++) {
        for (int c = 0; c < width; c++) {
            int i = r * width + c;
            uint8_t red = inPixels[i].x;
            uint8_t green = inPixels[i].y;
            uint8_t blue = inPixels[i].z;
            grayPixels[i] = 0.299f*red + 0.587f*green + 0.114f*blue;
        }
    }
}

__global__ void convertGrayscaleKernel(uchar3 *inPixels, int width, int height, uint8_t * grayPixels)
{
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height && c < width) {
		int i = r * width + c;
        uint8_t red = inPixels[i].x;
        uint8_t green = inPixels[i].y;
        uint8_t blue = inPixels[i].z;
        grayPixels[i] = 0.299f*red + 0.587f*green + 0.114f*blue;
    }
}

int abs(int x) {
    return x < 0 ? -1 * x : x;
}

void calcEnergy(uint8_t * grayPixels, int width, int height, int * energyMap,
                int * filterXSobel, int * filterYSobel, int filterWidth)
{
    for (int r = 0; r < height; r++) {
        for (int c = 0; c < width; c++) {
            int convolutionX = 0;
            int convolutionY = 0;

            for (int filterR = 0; filterR < filterWidth; filterR++) {
                for (int filterC = 0; filterC < filterWidth; filterC++) {
                    // Calc convolution with X-Sobel filter
                    int filterValX = filterXSobel[filterR * filterWidth + filterC];
                    int grayPixelsR = r - filterWidth / 2 + filterR;
                    int grayPixelsC = c - filterWidth / 2 + filterC;
                    grayPixelsR = min(max(0, grayPixelsR), height - 1);
                    grayPixelsC = min(max(0, grayPixelsC), width - 1);
                    uint8_t grayPixel = grayPixels[grayPixelsR * width + grayPixelsC];
                    convolutionX += filterValX * (int)grayPixel;

                    // Calc convolution with Y-Sobel filter
                    int filterValY = filterYSobel[filterR * filterWidth + filterC];
                    grayPixelsR = r - filterWidth / 2 + filterR;
                    grayPixelsC = c - filterWidth / 2 + filterC;
                    grayPixelsR = min(max(0, grayPixelsR), height - 1);
                    grayPixelsC = min(max(0, grayPixelsC), width - 1);
                    grayPixel = grayPixels[grayPixelsR * width + grayPixelsC];
                    convolutionY += filterValY * (int)grayPixel;
                }
            }

            energyMap[r * width + c] = abs(convolutionX) + abs(convolutionY);
        }
    }
}

__global__ void calcEnergyKernel(uint8_t * grayPixels, int width, int height, int * energyMap,
                                int * filterXSobel, int * filterYSobel, int filterWidth)
{
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height && c < width) {
        int i = r * width + c;

		int convolutionX = 0;
        int convolutionY = 0;

        for (int filterR = 0; filterR < filterWidth; filterR++) {
            for (int filterC = 0; filterC < filterWidth; filterC++) {
                // Calc convolution with X-Sobel filter
                int filterValX = filterXSobel[filterR * filterWidth + filterC];
                int grayPixelsR = r - filterWidth / 2 + filterR;
                int grayPixelsC = c - filterWidth / 2 + filterC;
                grayPixelsR = min(max(0, grayPixelsR), height - 1);
                grayPixelsC = min(max(0, grayPixelsC), width - 1);
                uint8_t grayPixel = grayPixels[grayPixelsR * width + grayPixelsC];
                convolutionX += filterValX * (int)grayPixel;

                // Calc convolution with Y-Sobel filter
                int filterValY = filterYSobel[filterR * filterWidth + filterC];
                grayPixelsR = r - filterWidth / 2 + filterR;
                grayPixelsC = c - filterWidth / 2 + filterC;
                grayPixelsR = min(max(0, grayPixelsR), height - 1);
                grayPixelsC = min(max(0, grayPixelsC), width - 1);
                grayPixel = grayPixels[grayPixelsR * width + grayPixelsC];
                convolutionY += filterValY * (int)grayPixel;
            }
        }

        energyMap[i] = abs(convolutionX) + abs(convolutionY);
    }
}

void findMinimumSeam(int * energyMap, int width, int height,
                    int * backtrack, int * L1, int * L2)
{
    memcpy(L1, energyMap, width * sizeof(int));

    for (int r = 1; r < height; r++) {
        for (int c = 0; c < width; c++) {
            int i = r * width + c;
            int idx;
            int energyMin = 1e9;
            
            for (int k = -1; k < 2; k++) {
                if ((c + k < 0) || (c + k == width))
                    continue;

                if (energyMin > L1[c + k]) {
                    energyMin = L1[c + k];
                    idx = k;
                }
            }

            backtrack[i] = c + idx;
            L2[c] = energyMap[i] + L1[c + idx];
        }
        memcpy(L1, L2, width * sizeof(int));
    }
}

__device__ int bCount = 0;
volatile __device__ int bCount1 = 0;

__global__ void findMinimumSeamKernel(int * energyMap, int width, int height,
                                    int * backtrack, int * L1, int * L2)
{
    __shared__ int bi;

    // Get the index bi that has the order
    if (threadIdx.x == 0) {
        bi = atomicAdd(&bCount, 1);
    }
    __syncthreads();

    int r = bi * blockDim.x / width;
    int c = bi * blockDim.x % width + threadIdx.x;

    if (r == 0) {
        if (c < width)
            L1[c] = energyMap[c];

        __syncthreads();

        if (threadIdx.x == 0)
            bCount1 += 1;
    }

    if (r > 0 && r < height) {
        while (bCount1 * blockDim.x / width < r) {}
    }

    if (r > 1) {
        if (c < width)
            L1[c] = L2[c];

        __syncthreads();

        if (threadIdx.x == 0 && (bCount1 + 1) * blockDim.x / width < r + 1) {
            bCount1 += 1;
        }
    }

    while (bCount1 < bi + 1 && (bCount1 + 1) * blockDim.x / width < r + 1) {}
    
    if (r > 0 && r < height && c < width) {
        int i = r * width + c;
        int idx;
        int energyMin = 1e9;
        
        for (int k = -1; k < 2; k++) {
            if ((c + k < 0) || (c + k == width))
                continue;

            if (energyMin > L1[c + k]) {
                energyMin = L1[c + k];
                idx = k;
            }
        }

        backtrack[i] = c + idx;
        L2[c] = energyMap[i] + L1[c + idx];
    }
    __syncthreads();

    if (threadIdx.x == 0 && (bCount1 + 1) * blockDim.x / width == r + 1) {
        bCount1 += 1;
    }
}

void deleteSeam(uchar3 * inPixels, int width, int height, uchar3 * outPixels,
                int * backtrack, int * L1, int * L2)
{
    int * seamPath = (int *)malloc(height * sizeof(int));
    int energyMin = 1e9, posMin;
    for (int i = 0; i < width; i++) {
        if (energyMin > L2[i]) {
            energyMin = L2[i];
            posMin = i;
        }
    }
    for (int r = height - 1; r >= 0; r--) {
        seamPath[r] = posMin;
        // printf("\n%i\n", seamPath[r]);
        posMin = backtrack[r * width + posMin];
    }

    for (int r = 0; r < height; r++) {
        for (int c = 0; c < width; c++) {
            int i = r * width + c;
            int _i = r * (width - 1) + c;
            if (c == seamPath[r])
                continue;
            if (c > seamPath[r])
                _i--;
            outPixels[_i] = inPixels[i];
        }
    }

    free(seamPath);
}

__global__ void deleteSeamKernel(uchar3 * inPixels, int width, int height, uchar3 * outPixels, int * seamPath)
{
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < height && c < width && c != seamPath[r]) {
        int i = r * width + c;
        int _i = r * (width - 1) + c;

        if (c > seamPath[r])
            _i--;

        outPixels[_i] = inPixels[i];
    }
}

void outputResult(uchar3 * dstPixels, int width, int height, uchar3 * srcPixels)
{
    for (int r = 0; r < height; r++) {
        for (int c = 0; c < width; c++) {
            int i = r * width + c;
            dstPixels[i] = srcPixels[i];
        }
    }
}

void seamCarving(uchar3 * inPixels, int width, int height, uchar3 * outPixels,
                int * filterXSobel, int * filterYSobel, int filterWidth,
                bool useDevice = false, dim3 blockSize = dim3(1, 1), int kernelType = 1)
{
    GpuTimer timer; 
    timer.Start();

    if (useDevice == false)
    {
        printf("\nSeam Carving by host\n");

        uint8_t * grayPixels = (uint8_t *)malloc(width * height * sizeof(uint8_t));
        int * energyMap = (int *)malloc(width * height * sizeof(int));
        int * backtrack = (int *)malloc(width * height * sizeof(int));
        int * L1 = (int *)malloc(width * sizeof(int));
        int * L2 = (int *)malloc(width * sizeof(int));

        uchar3 * tempPixels = (uchar3 *)malloc(width * height * sizeof(uchar3));
        memcpy(tempPixels, inPixels, width * height * sizeof(uchar3));

        for (int i = 0; i < 256; i++) {
            convertGrayscale(tempPixels, width - i, height, grayPixels);

            calcEnergy(grayPixels, width - i, height, energyMap, filterXSobel, filterYSobel, filterWidth);

            findMinimumSeam(energyMap, width - i, height, backtrack, L1, L2);

            deleteSeam(tempPixels, width - i, height, tempPixels, backtrack, L1, L2);
        }

        outputResult(outPixels, width - 256, height, tempPixels);

        free(grayPixels);
        free(energyMap);
        free(backtrack);
        free(L1);
        free(L2);
        free(tempPixels);
    }
    else // Use device
    {
        printf("\nSeam Carving by device\n");

        uint8_t * d_grayPixels;
        int * d_energyMap, * d_backtrack, * d_L1, * d_L2;
        uchar3 * d_tempPixels;

        int * d_filterXSobel, * d_filterYSobel;

        int * d_seamPath;

        CHECK(hipMalloc(&d_grayPixels, width * height * sizeof(uint8_t)));
        CHECK(hipMalloc(&d_energyMap, width * height * sizeof(int)));
        CHECK(hipMalloc(&d_backtrack, width * height * sizeof(int)));
        
        CHECK(hipMalloc(&d_L1, width * sizeof(int)));
        CHECK(hipMalloc(&d_L2, width * sizeof(int)));

        CHECK(hipMalloc(&d_tempPixels, width * height * sizeof(uchar3)));
        CHECK(hipMemcpy(d_tempPixels, inPixels, width * height * sizeof(uchar3), hipMemcpyHostToDevice));

        CHECK(hipMalloc(&d_filterXSobel, filterWidth * filterWidth * sizeof(int)));
        CHECK(hipMalloc(&d_filterYSobel, filterWidth * filterWidth * sizeof(int)));

        CHECK(hipMemcpy(d_filterXSobel, filterXSobel, filterWidth * filterWidth * sizeof(int), hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_filterYSobel, filterYSobel, filterWidth * filterWidth * sizeof(int), hipMemcpyHostToDevice));

        CHECK(hipMalloc(&d_seamPath, height * sizeof(int)));

        for (int i = 0; i < 256; i++) {
            dim3 gridSizeBlock2D((width - i - 1) / blockSize.x + 1, (height - 1) / blockSize.y + 1);
            int blockSize1D = (int)blockSize.x * (int)blockSize.y;
            dim3 gridSizeBlock1D(((width - i - 1) / blockSize1D + 1) * height);

            convertGrayscaleKernel<<<gridSizeBlock2D, blockSize>>>(d_tempPixels, width - i, height, d_grayPixels);
            hipDeviceSynchronize();
            CHECK(hipGetLastError());

            calcEnergyKernel<<<gridSizeBlock2D, blockSize>>>(d_grayPixels, width - i, height, d_energyMap,
                                                            d_filterXSobel, d_filterYSobel, filterWidth);
            hipDeviceSynchronize();
            CHECK(hipGetLastError());

            findMinimumSeamKernel<<<gridSizeBlock1D, blockSize1D>>>(d_energyMap, width - i, height,
                                                            d_backtrack, d_L1, d_L2);
            hipDeviceSynchronize();
            CHECK(hipGetLastError());


            int energyMin = 1e9;
            int * posMin = (int *)malloc(sizeof(int));
            int * curL2 = (int *)malloc(sizeof(int));
            
            for (int k = 0; k < width - i; k++) {
                CHECK(hipMemcpyFromSymbol(curL2, HIP_SYMBOL(&d_L2[k]), sizeof(int)));
                if (energyMin > curL2[0]) {
                    energyMin = curL2[0];
                    posMin[0] = k;
                }
            }
            for (int r = height - 1; r >= 0; r--) {
                CHECK(hipMemcpyToSymbol(HIP_SYMBOL(&d_seamPath[r]), posMin, sizeof(int)));
                // printf("\n%i\n", seamPath[r]);
                CHECK(hipMemcpyFromSymbol(posMin, HIP_SYMBOL(&d_backtrack[r * width + posMin[0]]), sizeof(int)));
            }

            free(posMin);
            free(curL2);


            deleteSeamKernel<<<gridSizeBlock2D, blockSize>>>(d_tempPixels, width - i, height, d_tempPixels, d_seamPath);
            hipDeviceSynchronize();
            CHECK(hipGetLastError());
        }

        CHECK(hipMemcpy(outPixels, d_tempPixels, (width - 256) * height * sizeof(uchar3), hipMemcpyDeviceToHost));

        CHECK(hipFree(d_grayPixels));
        CHECK(hipFree(d_energyMap));
        CHECK(hipFree(d_backtrack));
        CHECK(hipFree(d_L1));
        CHECK(hipFree(d_L2));
        CHECK(hipFree(d_tempPixels));
        CHECK(hipFree(d_filterXSobel));
        CHECK(hipFree(d_filterYSobel));
        CHECK(hipFree(d_seamPath));
    }

    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
}

float computeError(uchar3 *a1, uchar3 *a2, int n)
{
    float err = 0;
    for (int i = 0; i < n; i++)
    {
        err += abs((int)a1[i].x - (int)a2[i].x);
        err += abs((int)a1[i].y - (int)a2[i].y);
        err += abs((int)a1[i].z - (int)a2[i].z);
    }
    err /= (n * 3);
    return err;
}

void printError(uchar3 *deviceResult, uchar3 *hostResult, int width, int height)
{
    float err = computeError(deviceResult, hostResult, width * height);
    printf("Error: %f\n", err);
}

char *concatStr(const char *s1, const char *s2)
{
    char *result = (char *)malloc(strlen(s1) + strlen(s2) + 1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor);
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %lu bytes\n", devProv.totalGlobalMem);
    printf("CMEM: %lu bytes\n", devProv.totalConstMem);
    printf("L2 cache: %i bytes\n", devProv.l2CacheSize);
    printf("SMEM / one SM: %lu bytes\n", devProv.sharedMemPerMultiprocessor);

    printf("****************************\n");
}

int main(int argc, char **argv)
{
    if (argc != 3 && argc != 5)
    {
        printf("The number of arguments is invalid\n");
        return EXIT_FAILURE;
    }

    printDeviceInfo();

    // Read input image file
    int width, height;
    uchar3 *inPixels;
    readPnm(argv[1], width, height, inPixels);
    printf("\nImage size (width x height): %i x %i\n", width, height);

    // Set up a simple filter with blurring effect
    int filterWidth = FILTER_WIDTH;
    int *filterXSobel = (int *)malloc(filterWidth * filterWidth * sizeof(int));
    int *filterYSobel = (int *)malloc(filterWidth * filterWidth * sizeof(int));
    
    filterXSobel[0] = 1, filterXSobel[1] = 0, filterXSobel[2] = -1;
    filterXSobel[3] = 2, filterXSobel[4] = 0, filterXSobel[5] = -2;
    filterXSobel[6] = 1, filterXSobel[7] = 0, filterXSobel[8] = -1;

    filterYSobel[0] = 1, filterYSobel[1] = 2, filterYSobel[2] = 1;
    filterYSobel[3] = 0, filterYSobel[4] = 0, filterYSobel[5] = 0;
    filterYSobel[6] = -1, filterYSobel[7] = -2, filterYSobel[8] = -1;

    // Blur input image not using device
    uchar3 *outPixels = (uchar3 *)malloc((width - 256) * height * sizeof(uchar3));
    seamCarving(inPixels, width, height, outPixels, filterXSobel, filterYSobel, filterWidth);

    // Blur input image using device, kernel 1
    dim3 blockSize(32, 32); // Default
    if (argc == 5)
    {
        blockSize.x = atoi(argv[3]);
        blockSize.y = atoi(argv[4]);
    }
    uchar3 *outPixels1 = (uchar3 *)malloc((width - 256) * height * sizeof(uchar3));
    seamCarving(inPixels, width, height, outPixels, filterXSobel, filterYSobel, filterWidth, true, blockSize, 1);
    printError(outPixels1, outPixels, width - 256, height);

    // Write results to files
    char *outFileNameBase = strtok(argv[2], "."); // Get rid of extension 
    writePnm(outPixels, width - 256, height, concatStr(outFileNameBase, "_host.pnm"));
    writePnm(outPixels1, width - 256, height, concatStr(outFileNameBase, "_device1.pnm"));
    // writePnm(outPixels2, width, height, concatStr(outFileNameBase, "_device2.pnm"));
    // writePnm(outPixels3, width, height, concatStr(outFileNameBase, "_device3.pnm"));

    // Free memories
    free(inPixels);
    free(filterXSobel);
    free(filterYSobel);
    free(outPixels);
    free(outPixels1);
}
